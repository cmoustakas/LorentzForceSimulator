#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Include GLM
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>

#include <math.h>
/**
 * Φ = ∯E•dS 
 * Φ ≈ Σ{E•n}
 */


__global__ void matrix_multiplication( float *flux, float *vertices, float *normals, float *position, uint size, float E_field_factor){
    int row_index = blockIdx.x*blockDim.x + threadIdx.x;
    float sum = 0;
    uint coordinates = 3; /** R^3 = {x,y,z}*/
    const int ray = 1;
    float tmp_flux[768];

    if(row_index < size){
        tmp_flux[threadIdx.x] = 0;
        
        float x = vertices[row_index*coordinates]     ;//+ position[0];
        float y = vertices[row_index*coordinates + 1] ;//+ position[1];
        float z = vertices[row_index*coordinates + 2] ;//+ position[2];
        float len = x*x + z*z;
        if(len + y*y == ray && len != 0){
        
            float E_field[3] = {E_field_factor*x/(len),
                                0,
                                E_field_factor*z/(len)}; 

            for(uint it = 0; it < 3; it ++)
                tmp_flux[threadIdx.x] = E_field[it] * normals[row_index*coordinates + it] + tmp_flux[threadIdx.x];
        }
        
    }
    __syncthreads();
    if(!threadIdx.x){
        flux[blockIdx.x] = 0;
        float limit = (blockIdx.x < 3) ?  1.0f : 0.75f;
        
        for(uint it = 0; it++ ; it < (uint)(blockDim.x*limit) ){
            flux[blockIdx.x] += tmp_flux[it];            
        }
    }

}


void kernel_caller(float E_field_factor,
                   std::vector<glm::vec3> normal_vectors,
                   std::vector<glm::vec3> vertices,
                   glm::vec3 position_of_sphere)
{
    uint sz = normal_vectors.size();

    int num_of_threads;      
    int num_of_blocks;    

    hipOccupancyMaxPotentialBlockSize(&num_of_blocks, &num_of_threads, matrix_multiplication, 0, sz); 

    float *host_normals  = &(normal_vectors.at(0).x);
    float *host_vertices = &(vertices.at(0).x);
    float *host_position = &(position_of_sphere.x);
    float *host_flux = (float*)malloc(sizeof(float)*num_of_blocks);

    float *device_normals ; hipMalloc((void **)&device_normals,  sz);
    float *device_vertices; hipMalloc((void **)&device_vertices, sz);
    float *device_position; hipMalloc((void **)&device_position, 3);
    float *device_flux    ; hipMalloc((void **)&device_flux, num_of_blocks);



    hipMemcpy(device_normals , host_normals , sz, hipMemcpyHostToDevice); 
    hipMemcpy(device_vertices, host_vertices, sz, hipMemcpyHostToDevice);
    hipMemcpy(device_position, host_position, 3/*{x,y,z}*/ , hipMemcpyHostToDevice);


 
    //__global__ void matrix_multiplication( float *flux, float *vertices, float *normals, float *position, uint size, float E_field_factor){
    matrix_multiplication<<<num_of_blocks,num_of_threads>>>(
                                                          device_flux,
                                                          device_vertices,
                                                          device_normals,
                                                          device_position,
                                                          sz,
                                                          E_field_factor
                                                         );
    hipDeviceSynchronize();
    hipMemcpy(device_flux, host_flux, num_of_blocks, hipMemcpyDeviceToHost);
    float d_flux = 0.0f;
    for(int i=0;i<num_of_blocks;i++)
        d_flux = host_flux[i] + d_flux;
    std::cout << "GPU: " << d_flux << std::endl;


    
    glm::vec3 E;
    float flux = 0.0f;
    const int ray = 1;
    for(int i=0;i<sz;i++){
        
        float x = vertices.at(i).x;
        float y = vertices.at(i).y;
        float z = vertices.at(i).z;

        float len = x*x + z*z;
        if(len + y*y == ray && len != 0){ 
            
            E.x = E_field_factor*x/len;
            E.z = E_field_factor*z/len;
            flux =  E.z * normal_vectors.at(i).z + E.x * normal_vectors.at(i).x + flux ;
        }
    }

    std::cout << "CPU : " << flux  << std::endl;
    
    

}